#include "hip/hip_runtime.h"
#include "fix/fix-nonlinear-kernels-ansi.h"

namespace kaldi {
  namespace fix {
    template <typename Real>
    __global__ void _mapping(Real* out, const Real* in, const float x_rang, const int * x_arraybin, const int * y_arraybin, const int num_p, const float LB, const float UB, float amp, MatrixDim d, int src_stride) 
    {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      int j = blockIdx.y * blockDim.y + threadIdx.y;
      int dst_index = i + j * d.stride;
      int src_index = i + j * src_stride;
      if (i < d.cols && j < d.rows) {
        int pos = (int)((in[src_index] / (2*x_rang) + 0.5)*num_p);
        int order =  (int)((in[src_index] / x_rang + 1) * amp  + 0.5);
        if (pos < 0)
          out[dst_index]=LB;
        else if (pos >= num_p)
          out[dst_index]=UB;
        else {
          out[dst_index] = Real(((x_arraybin[pos+1]-order)*y_arraybin[pos]+(order-x_arraybin[pos])*y_arraybin[pos+1])/(x_arraybin[pos+1]-x_arraybin[pos])/amp);
        }
      }
    }

    void cuda_mapping(const dim3 dimGrid, const dim3 dimBlock, float* data, const float* in, const float x_rang, const int * x_arraybin, const int * y_arraybin, const int num_p, const float LB, const float UB, float amp, MatrixDim d, int src_stride) {
      _mapping <<<dimGrid, dimBlock>>>(data, in, x_rang, x_arraybin, y_arraybin, num_p, LB, UB, amp, d, src_stride);
    }
    void cuda_mapping(const dim3 dimGrid, const dim3 dimBlock, double* data, const double* in, const float x_rang, const int * x_arraybin, const int * y_arraybin, const int num_p, const float LB, const float UB, float amp, MatrixDim d, int src_stride) {
      _mapping <<<dimGrid, dimBlock>>>(data, in, x_rang, x_arraybin, y_arraybin, num_p, LB, UB, amp, d, src_stride);
    }
  }
}
